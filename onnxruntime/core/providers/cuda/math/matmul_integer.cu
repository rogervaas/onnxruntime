#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "matmul_integer.cuh"

#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

template <int TPB>
__global__ void ReduceSumOnLastAxisOneMatrix(const int8_t* a, int32_t* a_row_sum, const int8_t* offset, int32_t K) {
  int32_t thread_data = 0;
  const int8_t* a_row = a + blockIdx.x * K;
  for (int i = threadIdx.x; i < K; i += TPB) {
    thread_data += *(a_row + i);
  }

  using BlockReduce = hipcub::BlockReduce<int32_t, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int32_t sum = BlockReduce(temp_storage).Sum(thread_data);

  if (threadIdx.x == 0) {
    a_row_sum[blockIdx.x] = (*offset) * sum;
  }
}

Status ReduceSumOnLastAxis(const int8_t* a, int32_t* row_sum, const int8_t* offset, const MatMulComputeHelper& helper) {
  for (size_t batch = 0; batch < helper.OutputOffsets().size(); batch++) {
    ReduceSumOnLastAxisOneMatrix<static_cast<int>(GridDim::maxThreadsPerBlock)>
        <<<helper.M(), GridDim::maxThreadsPerBlock, 0>>>(a + helper.LeftOffsets()[batch],
                                                         row_sum + batch * helper.M(),
                                                         offset,
                                                         helper.K());
  }

  return Status::OK();
}

template <int TPB>
__global__ void ReduceSumOnSecondToLastAxisOneMatrix(const int8_t* b, int32_t* col_sum, const int8_t* offset, int32_t K, int32_t N) {
  int32_t thread_data = 0;
  const int8_t* col = b + blockIdx.x;
  for (int i = threadIdx.x; i < K; i += TPB) {
    thread_data += *(col + i * N);
  }

  using BlockReduce = hipcub::BlockReduce<int32_t, TPB>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int32_t sum = BlockReduce(temp_storage).Sum(thread_data);

  if (threadIdx.x == 0) {
    col_sum[blockIdx.x] = (*offset) * sum;
  }
}

Status ReduceSumOnSecondToLastAxis(const int8_t* b, int32_t* col_sum, const int8_t* offset, const MatMulComputeHelper& helper) {
  for (size_t batch = 0; batch < helper.OutputOffsets().size(); batch++) {
    ReduceSumOnSecondToLastAxisOneMatrix<static_cast<int>(GridDim::maxThreadsPerBlock)>
        <<<helper.N(), GridDim::maxThreadsPerBlock, 0>>>(b + helper.RightOffsets()[batch],
                                                         col_sum + batch * helper.N(),
                                                         offset,
                                                         helper.K(),
                                                         helper.N());
  }

  return Status::OK();
}

template <int TPB>
__global__ void InitializeMatrix(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, const int8_t* a_offset, const int8_t* b_offset, int32_t K, int32_t N) {
  for (int32_t i = threadIdx.x; i < N; i += TPB) {
    *(output + blockIdx.x * N + i) = K * (*a_offset) * (*b_offset) + row_sum[blockIdx.x] + col_sum[i];
  }
}

Status InitializeOutput(const int32_t* row_sum,
                        const int32_t* col_sum,
                        int32_t* output,
                        const int8_t* a_offset,
                        const int8_t* b_offset,
                        const MatMulComputeHelper& helper) {
  int blocksPerGrid = static_cast<int>(CeilDiv(helper.M() * helper.N(), GridDim::maxThreadsPerBlock));
  for (size_t batch = 0; batch < helper.OutputOffsets().size(); batch++) {
    InitializeMatrix<static_cast<int>(GridDim::maxThreadsPerBlock)>
        <<<helper.M(), GridDim::maxThreadsPerBlock, 0>>>(row_sum + batch * helper.M(),
                                                         col_sum + batch * helper.N(),
                                                         output + helper.OutputOffsets()[batch],
                                                         a_offset,
                                                         b_offset,
                                                         helper.K(),
                                                         helper.N());
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
